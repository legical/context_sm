#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include <iostream>
#include <thread>
#include <utility>

#include "util.cu"

using namespace std;

// __device__ void yesleep(float t) {
//     hipDeviceProp_t prop;
//     hipGetDeviceProperties(&prop, 0);
//     clock_t CLOCK_RATE = prop.clockRate;
//     clock_t t0 = clock64();
//     clock_t t1 = t0;
//     while ((t1 - t0)/(CLOCK_RATE*1000.0f) < t)
//         t1 = clock64();
// }

__global__ void MyKernel(int numSms, int numBlocks, int clockRate) {
    clock_t  start_clock = clock();
    float    Start_time = (float)start_clock / clockRate;
    uint32_t smid = getSMID();
    uint32_t blockid = getBlockIDInGrid();
    uint32_t threadid = getThreadIdInBlock();
    MySleep(35); // about 56ms

    clock_t end_clock = clock();
    float   End_time = (float)end_clock / clockRate;
    // if(i == 3)//只输出其中一个CPU线程的Kernel
    // printf("CPU thread %d:\t
    // SMID:%d,\tBlockID:%d,\tThreadID:%d,\tS_time:%f,\tE_time:%f\tSm_num:%d,\tBlock_num:%d!\n",i,smid,blockid,threadid,Start_time,End_time,numSms,numBlocks);
    if (blockid % 8 == 0) {
        printf(
            "Kernel need %d sms***\tSMID:%d,\tBlockID:%d,\tThreadID:%d,\tS_time:%f,\tE_time:%f\n",
            numSms, smid, blockid, threadid, Start_time, End_time);
    }

    return;
}

__global__ void Test_Kernel(int numBlocks, int numSms, int kernelID,
                            int clockRate) {
    // shared memory : 32 KB
    const uint32_t   SM_size = 32 * 1024 / sizeof(float);
    int              i = 0;
    __shared__ float s_tvalue[SM_size];
    clock_t          start_clock = clock();
    float            Start_time = (float)start_clock / clockRate;
    uint32_t         smid = getSMID();
    uint32_t         blockid = getBlockIDInGrid();
    uint32_t         threadid = getThreadIdInBlock();
    // MySleep(600);
    // clock_t end_clock = clock();
    // float   End_time = (float)end_clock / clockRate;

    for (i = 0; i < SM_size; i++) {
        s_tvalue[i] = i + 8;
    }
    while (i < SM_size) {
        i = s_tvalue[i];
    }
    clock_t end_clock = clock();
    float   End_time = (float)end_clock / clockRate;

    printf("%d\t%d\t%d\t%.6f\t%.6f\n", kernelID, blockid,
           smid, Start_time, End_time);
    return;
}

const char* MyGetRuntimeError(hipError_t error) {
    if (error != hipSuccess) {
        return hipGetErrorString(error);
    } else
        return NULL;
}

char* MyGetdeviceError(hipError_t error) {
    if (error != hipSuccess) {
        char* charerr = (char*)malloc(100);
        hipDrvGetErrorString(error, (const char**)&charerr);
        return charerr;
    } else
        return NULL;
}

int main(void) {
    //初始化
    // hipInit(0);
    int            device = 0;
    hipDeviceProp_t prop;
    const int      CONTEXT_POOL_SIZE = 4;
    hipCtx_t      contextPool[CONTEXT_POOL_SIZE];
    int            smCounts[CONTEXT_POOL_SIZE];
    hipSetDevice(device);
    // hipGetDevice(&device);
    // printf("device:%d\n",device);
    hipGetDeviceProperties(&prop, device);
    int clockRate = prop.clockRate;
    int sm_number = prop.multiProcessorCount;
    printf("*********   This GPU has %d SMs   *********\n", sm_number);

    smCounts[0] = 1;
    smCounts[1] = 2;
    smCounts[2] = (sm_number - 3) / 3;
    smCounts[3] = (sm_number - 3) / 3 * 2;

    //创建Context
    for (int i = 0; i < CONTEXT_POOL_SIZE; i++) {
        CUexecAffinityParam affinity;
        affinity.type = CU_EXEC_AFFINITY_TYPE_SM_COUNT;
        affinity.param.smCount.val = smCounts[i];

        hipError_t err2;
        err2 = cuCtxCreate_v3(&contextPool[i], &affinity, 1, 0, device);

        if (MyGetdeviceError(err2) != NULL) {
            printf("The %d cuCtxCreate_v3 Error:%s\n", i, MyGetdeviceError(err2));
        }
        // cuCtxCreate_v3
        // 创建带有affinity的上下文，并且CU_EXEC_AFFINITY_TYPE_SM_COUNT属性仅在Volta及更新的架构上以及MPS下可用
        //链接：https://docs.nvidia.com/cuda/cuda-driver-api/group__CUDA__CTX.html#group__CUDA__CTX_1g2a5b565b1fb067f319c98787ddfa4016
        // cuCtxCreate_v3(&contextPool[i], &affinity, 1, 0, deviceOrdinal);
    }

    std::thread mythread[CONTEXT_POOL_SIZE];
    int         step = 0;
    for (step = 0; step < CONTEXT_POOL_SIZE; step++)
        mythread[step] = std::thread([=]() {
            // printf("thread %d start!\n",i);
            int                 numSms = 0;
            int                 numBlocks = 0;
            int                 numBlocksPerSm = 0;
            int                 numThreads = 1; //每个Block中的Thread数目
            CUexecAffinityParam affinity;

            hipError_t err1;
            //将指定的CUDA上下文绑定到调用CPU线程
            err1 = hipCtxSetCurrent(contextPool[step]);
            if (err1 != hipSuccess) {
                printf("thread hipCtxSetCurrent Error:%s\n", MyGetdeviceError(err1));
            }

            hipError_t err2;
            // Returns the execution affinity setting for the current context
            err2 = cuCtxGetExecAffinity(&affinity, CU_EXEC_AFFINITY_TYPE_SM_COUNT);
            if (err2 != hipSuccess) {
                printf("thread cuCtxGetExecAffinity Error:%s\n",
                       MyGetdeviceError(err2));
            }

            //获取当前context对应的线程数目
            numSms = affinity.param.smCount.val;
            if (numSms != smCounts[step]) {
                printf("Context %d parititioning SM error!\tPlan:%d\tactual:%d\n", step,
                       smCounts[step], numSms);
                // cout<< "Context "<< step << " parititioning SM error!\tPlan:" <<
                // smCounts[step] << "\tactual:" << numSms << endl;
            } else {
                printf("Context %d parititioning SM successed!\tPlan:%d\tactual:%d\n", step,
                       smCounts[step], numSms);
            }
            // printf("numSms:%d\n",numSms);

            // hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, kernel,
            // numThreads, 0);
            //返回 Kernel的占用率
            hipError_t error1;
            error1 = hipOccupancyMaxActiveBlocksPerMultiprocessor(
                &numBlocksPerSm, MyKernel, numThreads, 0);
            if (error1 != hipSuccess) {
                printf(
                    "thread hipOccupancyMaxActiveBlocksPerMultiprocessor Error:%s\n",
                    MyGetRuntimeError(error1));
            }
            // printf("thread %d  numBlocksPerSm:%d\n",step, numBlocksPerSm);

            numBlocks = 1 + smCounts[step] * 17; // 2个SM，最多同时执行32个Block
            printf("KernelID\t%d\tSMnum\t%d\tBlocknum\t%d\n", step, numSms,
                   numBlocks);
            // cout << "KernelID\t"<< step << "\tSMnum\t" << numSms << "\tBlocknum\t"
            // << numBlocks << endl;
            // printf("Block nums:%d\n",numBlocks);
            dim3 dimBlock(numThreads, 1, 1); //每个Block中thread数目：numThreads
            dim3 dimGrid(numBlocks, 1, 1);   //每个Grid中Block数目

            printf("kernelID\tBlockID\tSMID\tStart_time\tEnd_time\n");

            Test_Kernel<<<dimGrid, dimBlock>>>(numBlocks, numSms, step, clockRate);
        });

    for (step = 0; step < CONTEXT_POOL_SIZE; step++)
        mythread[step].join();

    hipDeviceReset();
    return 0;
}
