#include "hip/hip_runtime.h"
/*
 * To compile for the read-only cache experiment (must explicitly specify
 * -arch-sm_xx since __ldg only works for GPU with cc>=35)
 *     Kepler: nvcc -arch=sm_35 -DRO fine_grain_texture_L1.cu -o test
 *     Pascal: nvcc -arch=sm_61 -DRO fine_grain_texture_L1.cu -o test
 * To compile fot the texture cache experiment
 *     nvcc -DTX fine_grain_texture_L1.cu -o test
 */

#include <stdio.h>
#include <stdlib.h>

//declare the texture
#if defined TX
texture<int, 1, hipReadModeElementType> tex_ref;
#elif defined RO
#include "hip/hip_runtime.h"
#endif
/*
 * We are going to traverse the array with a stride of _stride_
 * So the number of accesses in the array is N/_stride_,
 * which is the iterations of the P-chase loop.
 *
 * N is defined according to the cache size of the GPU.
 * stride is defined according to the stage:
 *   1 if determining cache size C and cache line size b
 *   b if determining cache set
 * ITER is defined as N/stride. Note that ITER will be used to
 * allocate shared memory. There are two methods:
 *   1. static shared memory
 *      In this method, ITER must be known at compile time.
 *      Therefore, __CUDA_ARCH__ is used to determine N and ITER
 *      according to the hardware CC.
 *      The author of the paper used 4096 as the ITER, i.e. they used
 *      a fixed number of iterations regardless of array size.
 *      This works when 4096 is larger than N/stride.
 *   2. Dymanic shared memory
 *      In this method, ITER can be computed by N, which can be
 *      defined either by __CUDA_ARCH__ or cudaDeviceProperty::major.
 *      But then we need to specify the shared memory size when
 *      launching the kernel and divided the shared memory into
 *      s_tvalue and s_value inside the kernel manually.
 *      E.g.
 *      texture_latency<<<1,1, ITER*2*sizeof(int)>>>( ... )
 *      Inside the kernel:
 *      extern __shared__ int s[];
 *      int *s_tvalue = s;
 *      int *s_value=&s_tvalue[ITER];
 *
 * We also need to pay attention to the size of shared memory.
 * The default shared memory size on Kepler (cc=3.5) and Pascal (cc=6.1)
 * is 48 KB, i.e. 12288 integers. When it is divided into 2 integer arrays,
 * each array can have maximum of 6144 integers, i.e. the maximum supported
 * ITER is 6144.
 * On Kepler, which has a texture cache of size 12 KB, shared memory is
 * large enough to hold tvalues and values for each iteration.
 * However, on Maxwell and Pascal, the shared memory is not able to hold
 * all values for each iteration.
 *
 * The above problem can be solved by warming up the cache first before
 * recording s_tvalue and s_value. Since the cache is warmed up, we only
 * need to go a few iterations to see the pattern, i.e. ITER = 256.
 *
 * Shared Memory (KB/SM)
 * ===================================================================================
 * Fermi           48/16 or 16/48 configurable with L1
 * Kepler(cc3.5)   48/16 or 16/48 or 32/32 configurable with L1
 * Kepler(cc3.7)   112/16 or 96/32 or 80/48 configurable with L1
 * Maxwell         96 dedicated shared memory
 * Pascall         64 dedicated shared memory (But one block can use up to 32 KB only)
 * Volta           96/32, 80/48, 64/64 configurable with L1
 * Turing
 * Ampere
 * ===================================================================================
 *
 */
// #if __CUDA_ARCH__ >= 500
// #define  N 6144
// #define  ITER 6144
// #else
// #define  N 3072
// #define  ITER 3072
// #endif

/*
 * The layout of textture cache and read-only cache is as follows:
 *
 * | Set 0   | Set 1   | Set 2   | Set 3   |
 * |---------|---------|---------|---------|
 * | line 0  | line 4  | line 8  | line 12 |
 * | line 1  | line 5  | line 9  | line 13 |
 * | line 2  | line 6  | line 10 | line 14 |
 * | line 3  | line 7  | line 11 | line 15 |
 * | line 16 | line 20 | line 24 | line 28 |
 * | line 17 | line 21 | line 25 | line 29 |
 * | line 18 | line 22 | line 26 | line 30 |
 * | line 19 | line 23 | line 27 | line 31 |
 * ....
 * When the replacement policy is LRU, one line miss in a set means evey line
 * in the same set will miss. If we want to see a pattern that every line
 * in set 0 will miss, we need to access at least 17 lines of data, i.e.
 * ITER >= 129.
 * Therefore, we choose ITER = 256 so that we won't moss such a pattern.
 */

/*
 * The layout of the read-only cache is different, since the memory addressing
 * is rather random.
 */
#define ITER 256

__global__ void cache_latency (
#if defined TX
    int * my_array,
#elif defined RO
    const int * __restrict__ my_array,
#endif
    int size,
    unsigned int *duration,
    int *index,
    int iter /* used to warm up the cache*/
    ) {

    // extern __shared__ int s[];
    // int *s_tvalue = s;
    // int *s_value=&s_tvalue[iter];

    const int it =  ITER;


   __shared__ unsigned int s_tvalue[it];
   __shared__ int s_value[it];

    unsigned int start, end;
    int i,j;

    //initilize j
    j=0;

    // for (i=0; i< iter; i++) {
    //     s_value[i] = -1;
    //     s_tvalue[i]=0;
    // }

    /*
     * Try to load the data first to avoid cold cache miss
     * Note that to warm up the cache, we need to traverse the whole array.
     */
    for (int cnt=0; cnt < iter; cnt++){
#if defined TX
        j=tex1Dfetch(tex_ref, j);
#elif defined RO
        j = __ldg(&my_array[j]);
#endif
    }
    /*
     * Since cold cache miss is avoided, the cache structure can
     * be explored now.
     */
    for (int cnt=0; cnt < it; cnt++) {
			
        start=clock();
#if defined TX
        j=tex1Dfetch(tex_ref, j);
#elif defined RO
        j = __ldg(&my_array[j]);
#endif
        s_value[cnt] = j;
			
        end=clock();
        s_tvalue[cnt] = (end -start);
    }

    for (i=0; i< it; i++){
	duration[i] = s_tvalue[i];
	index[i] = s_value[i];
    }

    // my_array[size] = i;
    // my_array[size+1] = s_tvalue[i-1];
}



void parametric_measure(int N, int stride) {
    // iterations=stride=1
    // N is the array size

    hipError_t error_id;

    int * h_a, * d_a;
    int size =  (N+2) * sizeof(int);
    h_a = (int *)malloc(size);
    //initialize array
    for (int i = 0; i < N; i++) {
        h_a[i] = (i + stride) % N;
    }
    h_a[N] = 0;
    h_a[N+1] = 0;
    hipMalloc ((void **) &d_a, size);
    //copy it to device array
    hipMemcpy((void *)d_a, (void *)h_a, size, hipMemcpyHostToDevice);

    // here to change the iteration numbers
    /*
     * We are going to traverse the array with a stride of _stride_
     * So the number of accesses in the array is N/_stride_,
     * which is the iterations of the P-chase loop.
     */
    /*
     * iterations is only used to traverse the array for warm up purpose
     * After warming up, we only keep ITER values.
     */
    int iterations = N/stride;
    int iter = ITER;

    // the time ivformation array and index array
    unsigned int *h_duration = (unsigned int *)malloc(iter*sizeof(unsigned int));
    int *h_index = (int *)malloc(iter*sizeof(int));
	
    int *d_index;
    error_id = hipMalloc(&d_index,  iter*sizeof(int));
    if (error_id != hipSuccess) {
        printf("Error 1.1 is %s\n", hipGetErrorString(error_id));
    }

    unsigned int *d_duration;
    error_id = hipMalloc(&d_duration,  iter*sizeof(unsigned int));
    if (error_id != hipSuccess) {
        printf("Error 1.2 is %s\n", hipGetErrorString(error_id));
    }


    //bind texture
#if defined TX
    hipBindTexture(0, tex_ref, d_a, size );
#endif

    hipDeviceSynchronize ();

    error_id = hipGetLastError();
    if (error_id != hipSuccess) {
        printf("Error 2 is %s\n", hipGetErrorString(error_id));
    }

    //for (int l=0; l < 20; l++) {

    // launch kernel
    dim3 Db = dim3(1);
    dim3 Dg = dim3(1,1,1);
    // texture_latency <<<Dg, Db, iter*2*sizeof(int)>>>(d_a, size, d_duration, d_index, iterations);
    cache_latency <<<Dg, Db>>>(d_a, size, d_duration, d_index, iterations);

    hipDeviceSynchronize ();

    error_id = hipGetLastError();
    if (error_id != hipSuccess) {
        printf("Error 3 is %s\n", hipGetErrorString(error_id));
    }

    hipDeviceSynchronize ();

    /* copy results from GPU to CPU */
    hipMemcpy((void *)h_index, (void *)d_index, iter*sizeof(int) , hipMemcpyDeviceToHost);
    hipMemcpy((void *)h_duration, (void *)d_duration, iter*sizeof(unsigned int) , hipMemcpyDeviceToHost);

    //}

    //print the result
    //printf("\n=====Visting the %f KB array, loop %d*%d times======\n", (float)(N)*sizeof(int)/1024.0f, iter, 1);
    for (int i=0;i<iter;i++){
        printf("%4d %10d\t %10f\n", i, h_index[i], (float)h_duration[i]);
    }


    //unbind texture
#ifdef TX
    hipUnbindTexture(tex_ref);
#endif

    //free memory on GPU
    hipFree(d_a);
    hipFree(d_duration);
    hipFree(d_index);
    hipDeviceSynchronize ();
	
    // free memory on CPU
    free(h_a);
    free(h_duration);
    free(h_index);
	
}

int main(int argc, char *argv[]) {

    int device = 0;
    hipSetDevice(device); // 0 for Kepler, 1 for Fermi
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);


    int stride, N;

    /*
     * Texture cache size (#integers) in each cc version
     */
    //int textureSize[7]={0,0,3072,3072,0,6144,6144};
    //txSize = textureSize[deviceProp.major];
    N = atoi(argv[1]);
    stride = atoi(argv[2]);

    //printf("\"%s\": cc=%d.%d, texture cache size=%d KB, shared memory=%ld KB\n",
    //       deviceProp.name, deviceProp.major, deviceProp.minor, N*4/1024, deviceProp.sharedMemPerBlock/1024);

    /*
     * The texture L1 data cache is
     * Fermi:   12 KB ==> 3072 integers
     * kepler:  12 KB ==> 3072 integers
     * Maxwell: 24 KB ==> 6144 integers
     * Pascal:  24 KB? ==> 6144 integers
     *
     * 1. To determine the cache size, N should start with a small value
     *    and increase to 3072. The first cache miss should appear when
     *    N=3073 ===> the cache size is 3072 integers = 12 KB
     *
     * 2. To determine the cache line size, N starts with 3073.
     *    The cache miss rate should stay close when 3073 <= N <= 3072+b
     *    So if we increase N gradually from 3073 and the first time we
     *    see a sudden increase on the cache miss, say N=3081, we can infer
     *    the cache line size b = 3081-3073 = 8 integers = 32 B
     */
    parametric_measure(N, stride);

    hipDeviceReset();
    //printf("\"%s\": cc=%d.%d, texture cache size=%d KB, shared memory=%ld KB\n",
    //       deviceProp.name, deviceProp.major, deviceProp.minor, N*4/1024, deviceProp.sharedMemPerBlock/1024);
    return 0;

}
