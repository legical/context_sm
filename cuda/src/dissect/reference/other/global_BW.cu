#include "hip/hip_runtime.h"
/*
 *
 * globalCopy.cu
 *
 * Microbenchmark for copy bandwidth of global memory.
 *
 * Build with: nvcc -I ../chLib <options> globalCopy.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "sys/time.h"

//SM number: 8(Fermi 560 Ti); 12(Kepler 780 ); 16 (Maxwell 980)
#define BLOCK_BASE (12)
#define MULTIPLIER (10)

template<const int n> 
__global__ void GlobalCopy(int *out, const int *in, size_t N )
{
    int temp[n];

	//avoid accessing cache, assure cold-cache access
	int start = n * blockIdx.x * blockDim.x + threadIdx.x;
	int step = n * blockDim.x * gridDim.x;
    
	int i;

    for ( i = start; i < N - step; i += step ) {
        for ( int j = 0; j < n; j++ ) {
            int index = i+j*blockDim.x;
            temp[j] = in[index];
        }
        for ( int j = 0; j < n; j++ ) {
            int index = i+j*blockDim.x;
            out[index] = temp[j];
        }
    }
    //there may be some elements left due to misaligning.
    for ( int j = 0; j < n; j++ ) {
        for ( int j = 0; j < n; j++ ) {
			int index = i + j*blockDim.x;
            if ( index<N ) temp[j] = in[index];
        }
        for ( int j = 0; j < n; j++ ) {
			int index = i + j*blockDim.x;
            if ( index<N ) out[index] = temp[j];
        }
    }
}

template<const int n>
double BandwidthCopy( int *deviceOut, int *deviceIn,
               int *hostOut, int *hostIn,
               size_t N,
               int cBlocks, int cThreads )
{
    double ret = 0.0;
    double elapsedTime;
    int cIterations;
    hipError_t status;

    for ( int i = 0; i < N; i++ ) {
        int r = rand();
		hostIn[i] = *(int *)(&r); // for small ints, LSBs; for int2 and int4, some stack cruft
    }

	memset(hostOut, 0, N*sizeof(int));
	hipMemcpy(deviceIn, hostIn, N*sizeof(int), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
    {
        // confirm that kernel launch with this configuration writes correct result
        GlobalCopy<n><<<cBlocks,cThreads>>>( 
            deviceOut,
            deviceIn,
            N );
		hipDeviceSynchronize();
		hipMemcpy(hostOut, deviceOut, N*sizeof(int), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
        status = hipGetLastError() ; 
		if (memcmp(hostOut, hostIn, (N)*sizeof(int))) {
            printf( "Incorrect copy performed!\n" );
            goto Error;
        }
    }

    cIterations = 10;
    struct timeval start, end;
    gettimeofday(&start,NULL);
	//time_t start, end;
	//start = clock();

    for ( int i = 0; i < cIterations; i++ ) {
		GlobalCopy<n> << <cBlocks, cThreads >> >(deviceOut, deviceIn, N);
    }

    hipDeviceSynchronize();
    // make configurations that cannot launch error-out with 0 bandwidth
	status = hipGetLastError();
    gettimeofday(&end,NULL);
    //end = clock();

    //elapsedTime =  (end - start)/1000.0;
    elapsedTime =  end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec)/1000000.0;

    // bytes per second
	ret = ((double)2 * N*cIterations*sizeof(int)) / elapsedTime;
    // gigabytes per second
    ret /= 1024.0*1048576.0;

Error:
    return ret;
}

template<const int n>
double ReportRow( size_t N, size_t threadStart, size_t threadStop, size_t cBlocks)
{
    int *deviceIn = 0;
	int *deviceOut = 0;
	int *hostIn = 0;
	int *hostOut = 0;

    hipError_t status;

    int maxThreads = 0;
    double maxBW = 0.0;

	hipMalloc(&deviceIn, N*sizeof(int));
	hipMalloc(&deviceOut, N*sizeof(int));
	hipMemset(deviceOut, 0, N*sizeof(int));

	hostIn = new int[N];
	hostOut = new int[N];
	if (!hostIn || !hostOut){
		if (hostIn) delete[] hostIn;
		if (hostOut) delete[] hostOut;

		hipFree(deviceIn);
		hipFree(deviceOut);
		return maxBW;
	}

    printf( "%d\t", n );

    for ( int cThreads = threadStart; cThreads <= threadStop; cThreads *= 2 ) {
        double bw = BandwidthCopy<n>(
            deviceOut, deviceIn, hostOut, hostIn, N,
             cBlocks, cThreads );
        if ( bw > maxBW ) {
            maxBW = bw;
            maxThreads = cThreads;
        }
        printf( "%.2f\t", bw );
    }
    printf( "%.2f\t%d\n", maxBW, maxThreads );

	delete[] hostIn;
	delete[] hostOut;

	hipFree(deviceIn);
	hipFree(deviceOut);
	return maxBW;
}

int main()
{
    int device = 0;
    int size = 64;
	int N = size * 1048576;
	int block_num;
	int threadStart = 32;
	int threadStop = 1024;

    printf( "Using coalesced reads and writes\n" );
	hipSetDevice(device);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	printf("\nDevice %d: \"%s\"\n", device, deviceProp.name);

	for ( block_num =int(BLOCK_BASE); block_num <= int(BLOCK_BASE * MULTIPLIER); block_num += int(BLOCK_BASE)){
		printf("\n=================Block number: %d=================\n", block_num);
		printf("Operand size: %d byte%c\n", sizeof(int), sizeof(int) == 1 ? '\0' : 's');
		printf("Input size: %dM operands\n", (int)(N >> 20));
		printf("                      Block Size\n");
		printf("Unroll\t");

		for (int cThreads = threadStart; cThreads <= threadStop; cThreads *= 2) {
			printf("%d\t", cThreads);
		}

		printf("maxBW\tmaxThreads\n");
		ReportRow<1>(N, threadStart, threadStop, block_num);
		ReportRow<2>(N, threadStart, threadStop, block_num);
		ReportRow<3>(N, threadStart, threadStop, block_num);
		ReportRow<4>(N, threadStart, threadStop, block_num);
		ReportRow<8>(N, threadStart, threadStop, block_num);

	}

    return 0;
}
