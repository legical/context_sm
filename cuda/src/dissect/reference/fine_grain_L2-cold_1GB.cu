#include "hip/hip_runtime.h"
#include "myutil.hpp"
#include "util.cuh"

//compile nvcc *.cu -o test

__global__ void global_latency (unsigned int * my_array, int array_length, int iterations,  unsigned int * duration, unsigned int *index);


void parametric_measure_global(int N, int iterations, int stride, char *filename);

void measure_global();

void GetFilename(char *filename)
{
    time_t timep;
    struct tm *p;

    time(&timep);          // 获取从1970至今过了多少秒，存入time_t类型的timep
    p = localtime(&timep); // 用localtime将秒数转化为struct tm结构体
    // 把格式化的时间写入字符数组中
    char path[96];
    getcwd(path, sizeof(path));
    // printf("2.2 dir__FILE__: %s\n", dirname(path));
    sprintf(filename, "%s/src/dissect/reference/data/L2_cache_data-%d%d%d.csv",
                dirname(path), p->tm_hour, p->tm_min, p->tm_sec);
}
int main(){

	hipSetDevice(0);

	measure_global();

	hipDeviceReset();
	return 0;
}


void measure_global() {

	int N, iterations, stride; 
	//stride in element
	iterations = 1;

    char *filename;
    filename = (char *)malloc(sizeof(char) * 256);
    GetFilename(filename);

	N = 1024 * 1024* 1024/sizeof(unsigned int); //in element
	for (stride = 1; stride <= N/2; stride*=2) {
		printf("\n=====%d GB array, cold cache miss, read 256 element====\n", N/1024/1024/1024);
		printf("Stride = %d element, %d bytes\n", stride, stride * sizeof(unsigned int));
		parametric_measure_global(N, iterations, stride ,filename);
		printf("===============================================\n\n");
	}
    
    free(filename);
}


void parametric_measure_global(int N, int iterations, int stride, char *filename) {
	hipDeviceReset();
	
	int i;
	unsigned int * h_a;
	/* allocate arrays on CPU */
	h_a = (unsigned int *)malloc(sizeof(unsigned int) * (N+2));
	unsigned int * d_a;
	/* allocate arrays on GPU */
	hipMalloc ((void **) &d_a, sizeof(unsigned int) * (N+2));

   	/* initialize array elements on CPU with pointers into d_a. */	
	for (i = 0; i < N; i++) {
	//original:	
		h_a[i] = (i+stride)%N;
	}

	h_a[N] = 0;
	h_a[N+1] = 0;
	/* copy array elements from CPU to GPU */
    hipMemcpy(d_a, h_a, sizeof(unsigned int) * N, hipMemcpyHostToDevice);

	unsigned int *h_index = (unsigned int *)malloc(sizeof(unsigned int)*256);
	unsigned int *h_timeinfo = (unsigned int *)malloc(sizeof(unsigned int)*256);

	unsigned int *duration;
	hipMalloc ((void **) &duration, sizeof(unsigned int)*256);

	unsigned int *d_index;
	hipMalloc( (void **) &d_index, sizeof(unsigned int)*256 );

	hipDeviceSynchronize ();
	/* launch kernel*/
	dim3 Db = dim3(1);
	dim3 Dg = dim3(1,1,1);
	global_latency <<<Dg, Db>>>(d_a, N, iterations,  duration, d_index);
	hipDeviceSynchronize ();

	hipError_t error_id = hipGetLastError();
        if (error_id != hipSuccess) {
		printf("Error kernel is %s\n", hipGetErrorString(error_id));
	}

	/* copy results from GPU to CPU */
	hipDeviceSynchronize ();
    hipMemcpy((void *)h_timeinfo, (void *)duration, sizeof(unsigned int)*256, hipMemcpyDeviceToHost);
    hipMemcpy((void *)h_index, (void *)d_index, sizeof(unsigned int)*256, hipMemcpyDeviceToHost);
	hipDeviceSynchronize ();\

    // 如果输出文件不存在，则创建文件并写入标题
    if (!isFileExists(filename))
    {
        // 读写文件。文件存在则追加写入，不存在则创建一个新文件
        FILE *fp = fopen(filename, "a+");
        // 如果打开文件失败
        if (fp == NULL)
        {
            // std::cout << "Can't open file : " << filename << std::endl;
            printf("Can't open file : %s \n", filename);
            fprintf(stderr, "fopen() failed.\n");
            exit(EXIT_FAILURE);
        }
        // 标题
        fprintf(fp, "Index,Exec_time,stride\n");
        fclose(fp);
    }

    // 读写文件。文件存在则追加写入，不存在则创建一个新文件
    FILE *fp = fopen(filename, "a+");
    // 如果打开文件失败
    if (fp == NULL)
    {
        // std::cout << "Can't open file : " << filename << std::endl;
        printf("Can't open file : %s \n", filename);
        fprintf(stderr, "fopen() failed.\n");
        exit(EXIT_FAILURE);
    }

	for(i=0;i<256;i++){
        fprintf(fp, "%d,%d,%d\n", h_index[i], h_timeinfo[i],stride);
        printf("%d\t %d\n", h_index[i], h_timeinfo[i]);
    }
	fclose(fp);	

	/* free memory on GPU */
	hipFree(d_a);
	hipFree(d_index);
	hipFree(duration);

    /*free memory on CPU */
    free(h_a);
    free(h_index);
	free(h_timeinfo);
	
	hipDeviceReset();
}

__global__ void global_latency (unsigned int * my_array, int array_length, int iterations, unsigned int * duration, unsigned int *index) {

	unsigned int start_time, end_time;
	unsigned int j = 0; 

	__shared__ unsigned int s_tvalue[256];
	__shared__ unsigned int s_index[256];

	int k;

	for(k=0; k<256; k++){
		s_index[k] = 0;
		s_tvalue[k] = 0;
	}

    /* for loop 256 times */
	for (k = 0; k < iterations*256; k++) {		
		start_time = clock();
		
		j = my_array[j];
		s_index[k]= j;
		end_time = clock();

        /* record execution time */
		s_tvalue[k] = end_time-start_time;
	}

	my_array[array_length] = j;
	my_array[array_length+1] = my_array[j];

	for(k=0; k<256; k++){
		index[k]= s_index[k];
		duration[k] = s_tvalue[k];
	}
}



