#include "hip/hip_runtime.h"
#include "myutil.hpp"
#include "util.cuh"

__global__ void read_random_arr(int *arr_gpu, const int ARR_SIZE/*, const int inter_cycle*/)
{
    uint32_t threadid = getThreadIdInBlock();
// #pragma unroll
    for (int j = 0; j < 3; j++)
    {
        int i = threadid;
#pragma unroll
        while (i < ARR_SIZE)
        {
            i = arr_gpu[i] + 31;
        }
        // for (int i = threadid; i < ARR_SIZE; i += 32)
        // {
        //     arr_gpu[i] |= i & 1;
        // }
    }
}

int main(int argc, char *argv[])
{
    // Default: array size = 1GB
    int ARR_SIZE = 1024 * 1024 * 256/*, inter_cycle = 8*/;

    int *arr, *arr_gpu;

    // allocate pinned memory in system memory
    gpuErrAssert(hipHostAlloc((void **)&arr,
                               ARR_SIZE * sizeof(int),
                               hipHostMallocDefault));
    init_chase_arr<int>(arr, ARR_SIZE, 1);

    gpuErrAssert(hipMalloc((void **)&arr_gpu, ARR_SIZE * sizeof(int)));

    // copy random memory from host to gpu
    gpuErrAssert(hipMemcpy(arr_gpu, arr, ARR_SIZE * sizeof(int), hipMemcpyHostToDevice));

    // run kernel for random GPU memory access
    read_random_arr<<<1, 32>>>(arr_gpu, ARR_SIZE/*, inter_cycle*/);

    // copy back random memory from gpu to host
    gpuErrAssert(hipMemcpy(arr, arr_gpu, ARR_SIZE * sizeof(int), hipMemcpyDeviceToHost));

    gpuErrAssert(hipFree(arr_gpu));
    gpuErrAssert(hipHostFree(arr));

    return 0;
}