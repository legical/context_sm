#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
int main(int argc, char *argv[])
{
    // printf("1. _pgmptr: %s\n", _pgmptr);

    char path[64];
    getcwd(path, sizeof(path));
    printf("1. getcwd: %s\n", path);

    printf("2. __FILE__: %s\n", __FILE__);

    printf("3. argv[0]: %s\n", argv[0]);
    
    return 0;
}