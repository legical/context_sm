
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MAX_DELAY 30

#include <time.h>
#include <sys/time.h>
#define USECPSEC 1000000ULL

unsigned long long dtime_usec(unsigned long long start){
  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

#define APPRX_CLKS_PER_SEC 1000000000ULL
__global__ void delay_kernel(unsigned seconds){
  unsigned long long dt = clock64();
  while (clock64() < (dt + (seconds*APPRX_CLKS_PER_SEC)));
}

int main(int argc, char *argv[]){
  unsigned delay_t = 10; // seconds, approximately
  // unsigned delay_t_r;
  // if (argc > 1) delay_t_r = atoi(argv[1]);
  // if ((delay_t_r > 0) && (delay_t_r < MAX_DELAY)) delay_t = delay_t_r;
  unsigned long long difft = dtime_usec(0);
  delay_kernel<<<1,1>>>(delay_t);
  hipDeviceSynchronize();
  difft = dtime_usec(difft);
  printf("kernel duration: %fs\n", difft/(float)USECPSEC);
  return 0;
}