#include "hip/hip_runtime.h"
# include <stdio.h>
# include <stdint.h>

# include "hip/hip_runtime.h"

//compile nvcc *.cu -o test

__global__ void global_latency (unsigned int * my_array, int array_length, int iterations,  unsigned int * duration, unsigned int *index);


void parametric_measure_global(int N, int iterations, int stride);

void measure_global();


int main(){

	hipSetDevice(0);

	measure_global();

	hipDeviceReset();
	return 0;
}


void measure_global() {

	int N, iterations, stride; 
	//stride in element
	iterations = 1;
	
	N = 1024 *4/sizeof(unsigned int); //in element
	for (stride = 1; stride <= 8; stride*=2) {
		printf("\n=====%d GB array, cold cache miss, read 256 element====\n", N/1024/1024/1024);
		printf("Stride = %d element, %d bytes\n", stride, stride * sizeof(unsigned int));
		parametric_measure_global(N, iterations, stride );
		printf("===============================================\n\n");
	}
}


void parametric_measure_global(int N, int iterations, int stride) {
	hipDeviceReset();
	
	int i;
	unsigned int * h_a;
	/* allocate arrays on CPU */
	h_a = (unsigned int *)malloc(sizeof(unsigned int) * (N+2));
	unsigned int * d_a;
	/* allocate arrays on GPU */
	hipMalloc ((void **) &d_a, sizeof(unsigned int) * (N+2));

   	/* initialize array elements on CPU with pointers into d_a. */
	
	for (i = 0; i < N; i++) {		
	//original:	
		h_a[i] = (i+stride)%N;	
	}

	h_a[N] = 0;
	h_a[N+1] = 0;
	/* copy array elements from CPU to GPU */
        hipMemcpy(d_a, h_a, sizeof(unsigned int) * N, hipMemcpyHostToDevice);

	unsigned int *h_index = (unsigned int *)malloc(sizeof(unsigned int)*4096);
	unsigned int *h_timeinfo = (unsigned int *)malloc(sizeof(unsigned int)*4096);

	unsigned int *duration;
	hipMalloc ((void **) &duration, sizeof(unsigned int)*4096);

	unsigned int *d_index;
	hipMalloc( (void **) &d_index, sizeof(unsigned int)*4096 );




	hipDeviceSynchronize ();
	/* launch kernel*/
	dim3 Db = dim3(1);
	dim3 Dg = dim3(1,1,1);


	global_latency <<<Dg, Db>>>(d_a, N, iterations,  duration, d_index);

	hipDeviceSynchronize ();

	hipError_t error_id = hipGetLastError();
        if (error_id != hipSuccess) {
		printf("Error kernel is %s\n", hipGetErrorString(error_id));
	}

	/* copy results from GPU to CPU */
	hipDeviceSynchronize ();



        hipMemcpy((void *)h_timeinfo, (void *)duration, sizeof(unsigned int)*4096, hipMemcpyDeviceToHost);
        hipMemcpy((void *)h_index, (void *)d_index, sizeof(unsigned int)*4096, hipMemcpyDeviceToHost);

	hipDeviceSynchronize ();

	for(i=0;i<4096;i++)
		printf("%d\t %d\n", h_index[i], h_timeinfo[i]);

	/* free memory on GPU */
	hipFree(d_a);
	hipFree(d_index);
	hipFree(duration);


        /*free memory on CPU */
        free(h_a);
        free(h_index);
	free(h_timeinfo);
	
	hipDeviceReset();	

}



__global__ void global_latency (unsigned int * my_array, int array_length, int iterations, unsigned int * duration, unsigned int *index) {

	unsigned int start_time, end_time;
	unsigned int j = 0; 

	__shared__ unsigned int s_tvalue[4096];
	__shared__ unsigned int s_index[4096];

	int k;

	for(k=0; k<4096; k++){
		s_index[k] = 0;
		s_tvalue[k] = 0;
	}

	for (k = 0; k < iterations*4096; k++) {
		
		start_time = clock();
		
		j = my_array[j];
		s_index[k]= j;
		end_time = clock();

		s_tvalue[k] = end_time-start_time;

	}

	my_array[array_length] = j;
	my_array[array_length+1] = my_array[j];

	for(k=0; k<4096; k++){
		index[k]= s_index[k];
		duration[k] = s_tvalue[k];
	}
}



