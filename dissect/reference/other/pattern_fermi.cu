#include "hip/hip_runtime.h"
# include <stdio.h>
# include <stdint.h>

# include "hip/hip_runtime.h"

//compile nvcc *.cu -o test

__global__ void global_latency (unsigned int * my_array, int array_length, int iterations,  unsigned int * duration, unsigned int *index);


void parametric_measure_global(int N, int iterations);

void measure_global();


int main(){

	hipSetDevice(1);

	measure_global();

	hipDeviceReset();
	return 0;
}


void measure_global() {

	int N, iterations; 
	//stride in element
	iterations = 1;
	
	N = 592*256*1024; 
		printf("\n=====%10.4f MB array, Fermi pattern read, read 160 element====\n", sizeof(unsigned int)*(float)N/1024/1024);
		parametric_measure_global(N, iterations);
		printf("===============================================\n\n");
	
}


void parametric_measure_global(int N, int iterations) {
	hipDeviceReset();

	hipError_t error_id;
	
	int i;
	unsigned int * h_a;
	/* allocate arrays on CPU */
	h_a = (unsigned int *)malloc(sizeof(unsigned int) * (N+2));
	unsigned int * d_a;
	/* allocate arrays on GPU */
	error_id = hipMalloc ((void **) &d_a, sizeof(unsigned int) * (N+2));
	if (error_id != hipSuccess) {
		printf("Error 1.0 is %s\n", hipGetErrorString(error_id));
	}


   	/* initialize array elements*/
	for (i=0; i<N; i++) 
		h_a[i] = 0;
	// 16MB*33 
	for (i=0; i<33; i++){ 
		h_a[i * 1024 * 256 * 16] = (i+1)*256*1024*16;
		h_a[i * 1024 * 256 * 16+1] = (1+i) * 1024 * 256 * 16+1;
		}
	// 1MB*63
	for (i=0; i<63 ; i++){
		h_a[(528+i)*256*1024] = (529+i)*256*1024;
	}
	h_a[528*256*1024+1] = 528*256*1024+2;
	h_a[528*256*1024+2] = 528*256*1024+3;
	h_a[528*256*1024+3] = 528*256*1024+1;
	h_a[591*256*1024 ] = 1;
	
	

	h_a[N] = 0;
	h_a[N+1] = 0;
	/* copy array elements from CPU to GPU */
        error_id = hipMemcpy(d_a, h_a, sizeof(unsigned int) * N, hipMemcpyHostToDevice);
	if (error_id != hipSuccess) {
		printf("Error 1.1 is %s\n", hipGetErrorString(error_id));
	}


	unsigned int *h_index = (unsigned int *)malloc(sizeof(unsigned int)*160);
	unsigned int *h_timeinfo = (unsigned int *)malloc(sizeof(unsigned int)*160);

	unsigned int *duration;
	error_id = hipMalloc ((void **) &duration, sizeof(unsigned int)*160);
	if (error_id != hipSuccess) {
		printf("Error 1.2 is %s\n", hipGetErrorString(error_id));
	}


	unsigned int *d_index;
	error_id = hipMalloc( (void **) &d_index, sizeof(unsigned int)*160 );
	if (error_id != hipSuccess) {
		printf("Error 1.3 is %s\n", hipGetErrorString(error_id));
	}





	hipDeviceSynchronize ();
	/* launch kernel*/
	dim3 Db = dim3(1);
	dim3 Dg = dim3(1,1,1);


	global_latency <<<Dg, Db>>>(d_a, N, iterations,  duration, d_index);

	hipDeviceSynchronize ();

	error_id = hipGetLastError();
        if (error_id != hipSuccess) {
		printf("Error kernel is %s\n", hipGetErrorString(error_id));
	}

	/* copy results from GPU to CPU */
	hipDeviceSynchronize ();



        error_id = hipMemcpy((void *)h_timeinfo, (void *)duration, sizeof(unsigned int)*160, hipMemcpyDeviceToHost);
	if (error_id != hipSuccess) {
		printf("Error 2.0 is %s\n", hipGetErrorString(error_id));
	}
        error_id = hipMemcpy((void *)h_index, (void *)d_index, sizeof(unsigned int)*160, hipMemcpyDeviceToHost);
	if (error_id != hipSuccess) {
		printf("Error 2.1 is %s\n", hipGetErrorString(error_id));
	}

	hipDeviceSynchronize ();

	for(i=0;i<160;i++)
		printf("%d\t %d\n", h_index[i], h_timeinfo[i]);

	/* free memory on GPU */
	hipFree(d_a);
	hipFree(d_index);
	hipFree(duration);


        /*free memory on CPU */
        free(h_a);
        free(h_index);
	free(h_timeinfo);
	
	hipDeviceReset();	

}



__global__ void global_latency (unsigned int * my_array, int array_length, int iterations, unsigned int * duration, unsigned int *index) {

	unsigned int start_time, end_time;
	unsigned int j = 0; 

	__shared__ unsigned int s_tvalue[160];
	__shared__ unsigned int s_index[160];

	int k;

	for(k=0; k<160; k++){
		s_index[k] = 0;
		s_tvalue[k] = 0;
	}

	//first round
//	for (k = 0; k < iterations*256; k++) 
//		j = my_array[j];
	
	//second round 
	for (k = 0; k < iterations*160; k++) {
		
			start_time = clock();

			j = my_array[j];
			s_index[k]= j;
			end_time = clock();

			s_tvalue[k] = end_time-start_time;

	}

	my_array[array_length] = j;
	my_array[array_length+1] = my_array[j];

	for(k=0; k<160; k++){
		index[k]= s_index[k];
		duration[k] = s_tvalue[k];
	}
}



