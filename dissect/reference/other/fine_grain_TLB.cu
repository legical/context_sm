#include "hip/hip_runtime.h"
# include <stdio.h>
# include <stdint.h>

# include "hip/hip_runtime.h"

//compile nvcc *.cu -o test

__global__ void global_latency (unsigned int * my_array, int array_length, int iterations,  unsigned int * duration, unsigned int *index);


void parametric_measure_global(int N, int iterations, int stride);

void measure_global();


int main(){

	hipSetDevice(0);

	measure_global();

	hipDeviceReset();
	return 0;
}


void measure_global() {

	int N, iterations, stride; 
	//stride in element
	iterations = 1;
	
	stride = 2048*1024/sizeof(unsigned int); //2MB stride
	//1. The L1 TLB has 16 entries. Test with N_min=28 *1024*256, N_max>32*1024*256
	//2. The L2 TLB has 65 entries. Test with N_min=128*1024*256, N_max=160*1024*256
	for (N = 28*1024*256; N <= 46*1024*256; N+=stride) {
		printf("\n=====%3.1f MB array, warm TLB, read 256 element====\n", sizeof(unsigned int)*(float)N/1024/1024);
		printf("Stride = %d element, %d MB\n", stride, stride * sizeof(unsigned int)/1024/1024);
		parametric_measure_global(N, iterations, stride );
		printf("===============================================\n\n");
	}
}


void parametric_measure_global(int N, int iterations, int stride) {
	hipDeviceReset();

	hipError_t error_id;
	
	int i;
	unsigned int * h_a;
	/* allocate arrays on CPU */
	h_a = (unsigned int *)malloc(sizeof(unsigned int) * (N+2));
	unsigned int * d_a;
	/* allocate arrays on GPU */
	error_id = hipMalloc ((void **) &d_a, sizeof(unsigned int) * (N+2));
	if (error_id != hipSuccess) {
		printf("Error 1.0 is %s\n", hipGetErrorString(error_id));
	}


   	/* initialize array elements on CPU with pointers into d_a. */
	
	for (i = 0; i < N; i++) {		
	//original:	
		h_a[i] = (i+stride)%N;	
	}

	h_a[N] = 0;
	h_a[N+1] = 0;
	/* copy array elements from CPU to GPU */
        error_id = hipMemcpy(d_a, h_a, sizeof(unsigned int) * N, hipMemcpyHostToDevice);
	if (error_id != hipSuccess) {
		printf("Error 1.1 is %s\n", hipGetErrorString(error_id));
	}


	unsigned int *h_index = (unsigned int *)malloc(sizeof(unsigned int)*256);
	unsigned int *h_timeinfo = (unsigned int *)malloc(sizeof(unsigned int)*256);

	unsigned int *duration;
	error_id = hipMalloc ((void **) &duration, sizeof(unsigned int)*256);
	if (error_id != hipSuccess) {
		printf("Error 1.2 is %s\n", hipGetErrorString(error_id));
	}


	unsigned int *d_index;
	error_id = hipMalloc( (void **) &d_index, sizeof(unsigned int)*256 );
	if (error_id != hipSuccess) {
		printf("Error 1.3 is %s\n", hipGetErrorString(error_id));
	}





	hipDeviceSynchronize ();
	/* launch kernel*/
	dim3 Db = dim3(1);
	dim3 Dg = dim3(1,1,1);


	global_latency <<<Dg, Db>>>(d_a, N, iterations,  duration, d_index);

	hipDeviceSynchronize ();

	error_id = hipGetLastError();
        if (error_id != hipSuccess) {
		printf("Error kernel is %s\n", hipGetErrorString(error_id));
	}

	/* copy results from GPU to CPU */
	hipDeviceSynchronize ();



        error_id = hipMemcpy((void *)h_timeinfo, (void *)duration, sizeof(unsigned int)*256, hipMemcpyDeviceToHost);
	if (error_id != hipSuccess) {
		printf("Error 2.0 is %s\n", hipGetErrorString(error_id));
	}
        error_id = hipMemcpy((void *)h_index, (void *)d_index, sizeof(unsigned int)*256, hipMemcpyDeviceToHost);
	if (error_id != hipSuccess) {
		printf("Error 2.1 is %s\n", hipGetErrorString(error_id));
	}

	hipDeviceSynchronize ();

	for(i=0;i<256;i++)
		printf("%d\t %d\n", h_index[i], h_timeinfo[i]);

	/* free memory on GPU */
	hipFree(d_a);
	hipFree(d_index);
	hipFree(duration);


        /*free memory on CPU */
        free(h_a);
        free(h_index);
	free(h_timeinfo);
	
	hipDeviceReset();	

}



__global__ void global_latency (unsigned int * my_array, int array_length, int iterations, unsigned int * duration, unsigned int *index) {

	unsigned int start_time, end_time;
	unsigned int j = 0; 

	__shared__ unsigned int s_tvalue[256];
	__shared__ unsigned int s_index[256];

	int k;

	for(k=0; k<256; k++){
		s_index[k] = 0;
		s_tvalue[k] = 0;
	}

	//first round, warm the TLB
	for (k = 0; k < iterations*256; k++) 
		j = my_array[j];
	
	//second round, begin timestamp  
	for (k = 0; k < iterations*256; k++) {
		
		start_time = clock();
		
		j = my_array[j];
		s_index[k]= j;
		end_time = clock();

		s_tvalue[k] = end_time-start_time;

	}

	my_array[array_length] = j;
	my_array[array_length+1] = my_array[j];

	for(k=0; k<256; k++){
		index[k]= s_index[k];
		duration[k] = s_tvalue[k];
	}
}



